/*
 *  Copyright 2008-2009 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include <iostream>
#include <stdio.h>
#include "cmdline.h"
#include "sparse_io.h"
#include "sparse_formats.h"
#include "test_spmm.h"
#include "benchmark_ell.h"
#include "spmm_ell_device.cu.h"

template <typename IndexType, typename ValueType>
void test_ell_matrix_kernel(const csr_matrix<IndexType,ValueType>& csr)
{
 
   //Test the performance of ell kernel
   benchmark_ell_on_device(csr, spmm_ell_device<IndexType, ValueType>,"ell");

}

template <typename IndexType, typename ValueType>
void run_ell(int argc, char **argv)
{
    char * mm_filename = NULL;
    for(int i = 1; i < argc; i++){
        if(argv[i][0] != '-'){
            mm_filename = argv[i];
            break;
        }
    }
    

    csr_matrix<IndexType,ValueType> csr;

    csr= read_csr_matrix<IndexType,ValueType>(mm_filename);
            

    printf("Using %d-by-%d matrix with %d nonzero values\n", csr.num_rows, csr.num_cols, csr.num_nonzeros); 

    // fill matrix with random values: some matrices have extreme values, 
    // which makes correctness testing difficult, especially in single precision
    srand(13);
    for(IndexType i = 0; i < csr.num_nonzeros; i++){
      csr.Ax[i] = 1.0 - 2.0 * (rand() / (RAND_MAX + 1.0)); 
    }
    
    // Call the function that tests the correctness and performance of ell kernel
    test_ell_matrix_kernel(csr);
    
    delete_host_matrix(csr);
}

int main(int argc, char** argv)
{
    int precision = 64;
    char * precision_str = get_argval(argc, argv, "precision");
    if(precision_str != NULL)
        precision = atoi(precision_str);
    printf("\nUsing %d-bit floating point precision\n\n", precision);

    if(precision ==  32){
        run_ell<unsigned int, float>(argc,argv);
    }
    else if(precision == 64)
    {
        int current_device = -1;
        hipDeviceProp_t properties;
        hipGetDevice(&current_device);
        hipGetDeviceProperties(&properties, current_device);
        if (properties.major == 1 && properties.minor < 3)
            std::cerr << "ERROR: Support for \'double\' requires Compute Capability 1.3 or greater\n\n";
        else
        run_ell<unsigned int, double>(argc,argv);
    }
   
    return EXIT_SUCCESS;
}

